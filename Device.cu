#include "Device.cuh"
#include <iostream>

namespace Device{

	hipblasHandle_t cublasHandle;

	void init() {
		hipError_t hipError_t(hipSetDevice(0));
		if (hipError_t != hipSuccess) {
			throw std::runtime_error("Device::init() failed to initialize device.");
		}
		hipblasStatus_t status(hipblasCreate(&cublasHandle));
		if (status != HIPBLAS_STATUS_SUCCESS) {
			if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
				throw std::runtime_error("Device::init() failed to initialize cuBLAS.");
			}
			else if(status == HIPBLAS_STATUS_ALLOC_FAILED) {
				throw std::runtime_error("Device::init() failed to allocate resources for cuBLAS.");
			}
		}
	}
	void reset() {
		hipblasStatus_t status(hipblasDestroy(cublasHandle));
		if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
			throw std::runtime_error("Device::init() failed to reset device? (cuBLAS library was not initialized)");
		}
		hipError_t hipError_t(hipDeviceReset());
		if (hipError_t != hipSuccess) {
			throw std::runtime_error("Device::reset() failed to reset device.");
		}
	}
}